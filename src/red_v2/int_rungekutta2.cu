#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "ode.h"
#include "int_rungekutta2.h"

#include "redutil2.h"
#include "macro.h"

// The Runge-Kutta matrix
var_t int_rungekutta2::a[] = 
{ 
	0.0,     0.0, 
	1.0/2.0, 0.0
};
// weights
var_t int_rungekutta2::b[] = { 0.0, 1.0     };
// nodes
var_t int_rungekutta2::c[] = { 0.0, 1.0/2.0 };

__constant__ var_t dc_a[ sizeof(int_rungekutta2::a ) / sizeof(var_t)];
__constant__ var_t dc_b[ sizeof(int_rungekutta2::b ) / sizeof(var_t)];
__constant__ var_t dc_c[ sizeof(int_rungekutta2::c ) / sizeof(var_t)];


namespace rk2_kernel
{
// a_i = b_i + F * c_i
static __global__
	void calc_lin_comb(var_t* a, const var_t* b, var_t F, const var_t* c, uint32_t n)
{
	uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t stride = gridDim.x * blockDim.x;

	while (n > tid)
	{
		a[tid] = b[tid] + F * c[tid];
		tid += stride;
	}
}

static __global__
	void calc_lin_comb(uint32_t n, uint32_t offset, var_t dt, const var_t *y_n, var_t** dydt, var_t *y_np1)
{
	const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < n)
	{
		y_np1[tid] = y_n[tid];
		for (int i = 0; i < 11; i++)
		{
			if (0.0 == dc_b[i])
			{
				continue;
			}
			y_np1[tid] += dt * dc_b[i] * dydt[offset + i][tid];
		}
	}
}
} /* namespace rk2_kernel */

int_rungekutta2::int_rungekutta2(ode& f, var_t dt, comp_dev_t comp_dev) :
	integrator(f, dt, false, 0.0, 2, comp_dev)
{
	name    = "Runge-Kutta2";
	n_order = 2;

	if (COMP_DEV_GPU == comp_dev)
	{
		redutil2::copy_constant_to_device(dc_a, a, sizeof(a));
		redutil2::copy_constant_to_device(dc_b, b, sizeof(b));
		redutil2::copy_constant_to_device(dc_c, c, sizeof(c));
	}
}

int_rungekutta2::~int_rungekutta2()
{}

void int_rungekutta2::calc_lin_comb(var_t* a, const var_t* b, var_t F, const var_t* c, uint32_t n)
{
	for (uint32_t tid = 0; tid < n; tid++)
	{
		a[tid] = b[tid] + F * c[tid];
	}
}

void int_rungekutta2::calc_y_np1()
{
	if (COMP_DEV_GPU == comp_dev)
	{
		rk2_kernel::calc_lin_comb<<<grid, block>>>(f.yout, f.y, dt_try, k[1], f.n_var);
		CUDA_CHECK_ERROR();
	}
	else
	{
		calc_lin_comb(f.yout, f.y, dt_try, k[1], f.n_var);
	}
}

void int_rungekutta2::calc_ytemp(uint16_t stage)
{
	if (COMP_DEV_GPU == comp_dev)
	{
	}
	else
	{
		for (uint32_t i = 0; i < f.n_var; i++)
		{
			var_t dy = 0.0;
			for (uint16_t j = 0; j < stage; j++)
			{
				dy += a[stage * n_stage + j] * k[j][i];
			}
			ytemp[i] = f.y[i] + dt_try * dy;
		}
	}
}

var_t int_rungekutta2::step()
{
	if (COMP_DEV_GPU == comp_dev)
	{
		redutil2::set_kernel_launch_param(f.n_var, THREADS_PER_BLOCK, grid, block);
	}

	uint16_t stage = 0;
	t = f.t;
	// Calculate initial differentials and store them into h_k
	f.calc_dy(stage, t, f.y, k[stage]);

	stage = 1;
	t = f.t + c[stage] * dt_try;
	calc_ytemp(stage);
	f.calc_dy(stage, t, ytemp, k[stage]);

	calc_y_np1();

	dt_did = dt_try;

	update_counters(1);

	f.tout = t = f.t + dt_did;
	f.swap();

	return dt_did;
}

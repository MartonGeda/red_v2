#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "ode.h"
#include "int_rungekutta2.h"

#include "redutil2.h"
#include "macro.h"

using namespace redutil2;

// The Runge-Kutta matrix
var_t int_rungekutta2::a[] = 
{ 
	0.0,     0.0,      // y = yn          -> k1
	1.0/2.0, 0.0       // y = yn + h*k1
};
// weights
var_t int_rungekutta2::b[] = { 0.0, 1.0     };
// nodes
var_t int_rungekutta2::c[] = { 0.0, 1.0/2.0 };
// These arrays will contain the stepsize multiplied by the constants
var_t int_rungekutta2::_a[ sizeof(int_rungekutta2::a ) / sizeof(var_t)];
var_t int_rungekutta2::_b[ sizeof(int_rungekutta2::b ) / sizeof(var_t)];
var_t int_rungekutta2::_c[ sizeof(int_rungekutta2::c ) / sizeof(var_t)];

__constant__ var_t dc_a[ sizeof(int_rungekutta2::a ) / sizeof(var_t)];


int_rungekutta2::int_rungekutta2(ode& f, comp_dev_t comp_dev) :
	integrator(f, false, 0.0, 2, comp_dev)
{
	name    = "Runge-Kutta2";
	n_order = 2;
}

int_rungekutta2::~int_rungekutta2()
{ }

void int_rungekutta2::calc_ytemp(uint16_t stage)
{
	if (PROC_UNIT_GPU == comp_dev.proc_unit)
	{
		var_t* coeff = dc_a + stage * n_stage;
		gpu_calc_lin_comb_s(ytemp, f.y, d_k, coeff, stage, f.n_var, comp_dev.id_dev, optimize);
	}
	else
	{
		var_t* coeff = _a + stage * n_stage;
		tools::calc_lin_comb_s(ytemp, f.y, h_k, coeff, stage, f.n_var);
	}
}

void int_rungekutta2::calc_y_np1()
{
	if (PROC_UNIT_GPU == comp_dev.proc_unit)
	{
		gpu_calc_lin_comb_s(f.yout, f.y, k[1], dt_try, f.n_var, comp_dev.id_dev, optimize);
	}
	else
	{
		tools::calc_lin_comb_s(f.yout, f.y, k[1], dt_try, f.n_var);
	}
}

var_t int_rungekutta2::step()
{
	static const uint16_t n_a = sizeof(int_rungekutta2::a) / sizeof(var_t);
	static uint32_t n_var = 0;

    if (n_var != f.n_var)
	{
		optimize = true;
		n_var = f.n_var;
	}
	else
	{
		optimize = false;
	}

	uint16_t stage = 0;
	t = f.t;
	// Calculate initial differentials and store them into h_k
	f.calc_dy(stage, t, f.y, k[stage]);

	// TODO: check if this speeds up the app or not!
	// Compute in advance the dt_try * coefficients to save n_var multiplication per stage
	for (uint16_t i = 0; i < n_a; i++)
	{
		_a[i] = dt_try * a[i];
	}
	if (PROC_UNIT_GPU == comp_dev.proc_unit)
	{
		redutil2::copy_constant_to_device(dc_a, _a, sizeof(_a));
	}

	stage = 1;
	t = f.t + c[stage] * dt_try;
	calc_ytemp(stage);
	f.calc_dy(stage, t, ytemp, k[stage]);

	calc_y_np1();

	dt_did = dt_try;
	f.tout = t = f.t + dt_did;
	f.swap();

    update_counters(1);

	return dt_did;
}

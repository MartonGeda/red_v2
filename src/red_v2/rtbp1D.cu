#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>

#include "rtbp1D.h"

#include "redutil2.h"
#include "constants.h"

using namespace std;
using namespace redutil2;


rtbp1D::rtbp1D(uint16_t n_ppo, comp_dev_t comp_dev) :
	//ode(1, 1, 3, n_ppo, comp_dev)
	ode(1, 1, 2, n_ppo, comp_dev)
{
	initialize();
	allocate_storage();
}

rtbp1D::~rtbp1D()
{
	deallocate_storage();
}

void rtbp1D::initialize()
{
	h_md       = 0x0;

	integral.h = 0.0;            // energy
	h_y[2]     = 0.0;            // t_0: real time
}

void rtbp1D::allocate_storage()
{
	allocate_host_storage();
	if (COMP_DEV_GPU == comp_dev)
	{
		allocate_device_storage();
	}
}

void rtbp1D::allocate_host_storage()
{
	ALLOCATE_HOST_VECTOR((void**)&(h_md),    n_obj * sizeof(tbp1D_t::metadata_t));
}

void rtbp1D::allocate_device_storage()
{
	ALLOCATE_DEVICE_VECTOR((void**)&(d_md),    n_obj * sizeof(tbp1D_t::metadata_t));
}

void rtbp1D::deallocate_storage()
{
	deallocate_host_storage();
	if (COMP_DEV_GPU == comp_dev)
	{
		deallocate_device_storage();
	}
}

void rtbp1D::deallocate_host_storage()
{
	FREE_HOST_VECTOR((void **)&(h_md));
}

void rtbp1D::deallocate_device_storage()
{
	FREE_DEVICE_VECTOR((void **)&(d_md));
}

void rtbp1D::trans_to_descartes_var(var_t& x, var_t& vx)
{
	x  = SQR(h_y[0]);
	vx = (2.0/h_y[0]) * (h_y[0] * h_y[1]);
}

void rtbp1D::calc_integral()
{
	const tbp1D_t::param_t* p = (tbp1D_t::param_t*)h_p;

	integral.h = (2.0 * SQR(h_y[1]) - p[0].mu ) / SQR(h_y[0]);
}

void rtbp1D::calc_dy(uint16_t stage, ttt_t curr_t, const var_t* y_temp, var_t* dy)
{
	if (COMP_DEV_CPU == comp_dev)
	{
		cpu_calc_dy(stage, curr_t, y_temp, dy);
	}
	else
	{
		gpu_calc_dy(stage, curr_t, y_temp, dy);
	}
}

void rtbp1D::cpu_calc_dy(uint16_t stage, ttt_t curr_t, const var_t* y_temp, var_t* dy)
{
	dy[0] = y_temp[1];                         // dy1 / ds = y2

	dy[1] = (integral.h / 2.0) * y_temp[0];    // dy2 / ds = h/2 * y1

	dy[2] = SQR(y_temp[0]);                    // dy3 / ds = y1^2
}

void rtbp1D::gpu_calc_dy(uint16_t stage, ttt_t curr_t, const var_t* y_temp, var_t* dy)
{
	throw string("The gpu_calc_dy() is not implemented.");
}

void rtbp1D::load(string& path)
{
	ifstream input;

	cout << "Loading " << path << " ";

	data_rep_t repres = (file::get_extension(path) == "txt" ? DATA_REPRESENTATION_ASCII : DATA_REPRESENTATION_BINARY);
	switch (repres)
	{
	case DATA_REPRESENTATION_ASCII:
		input.open(path.c_str());
		if (input) 
		{
			load_ascii(input);
		}
		else 
		{
			throw string("Cannot open " + path + ".");
		}
		break;
	case DATA_REPRESENTATION_BINARY:
		input.open(path.c_str(), ios::binary);
		if (input) 
		{
			load_binary(input);
		}
		else 
		{
			throw string("Cannot open " + path + ".");
		}
		break;
	}
	input.close();

	cout << " done" << endl;
}

void rtbp1D::load_ascii(ifstream& input)
{
	tbp1D_t::param_t* p = (tbp1D_t::param_t*)h_p;

	for (uint32_t i = 0; i < n_obj; i++)
	{
		load_ascii_record(input, &t, &h_md[i], &p[i], &h_y[i], &h_y[i+1]);
	}
}

void rtbp1D::load_ascii_record(ifstream& input, ttt_t* _t, tbp1D_t::metadata_t *md, tbp1D_t::param_t* p, var_t* x, var_t* vx)
{
	string name;

	// epoch
	input >> *_t;
	// id
	input >> md->id;
	// mu = k^2*(m1 + m2)
	input >> p->mu;
	// position
	input >> *x;
	// velocity
	input >> *vx;
}

void rtbp1D::load_binary(ifstream& input)
{
	throw string("The load_binary() is not implemented.");
}

void rtbp1D::print_solution(std::string& path_si, std::string& path_sd, data_rep_t repres)
{
	ofstream sout;

	switch (repres)
	{
	case DATA_REPRESENTATION_ASCII:
		sout.open(path_si.c_str(), ios::out | ios::app);
		if (sout)
		{
			print_solution_ascii(sout);
		}
		else
		{
			throw string("Cannot open " + path_si + ".");
		}
		break;
	case DATA_REPRESENTATION_BINARY:
		sout.open(path_si.c_str(), ios::out | ios::app | ios::binary);
		if (sout)
		{
			print_solution_binary(sout);
		}
		else
		{
			throw string("Cannot open " + path_si + ".");
		}
		break;
	default:
		throw string("Parameter 'repres' is out of range.");
	}
	sout.close();
}

void rtbp1D::print_solution_ascii(ofstream& sout)
{
	sout.precision(16);
	sout.setf(ios::right);
	sout.setf(ios::scientific);

	var_t x  = 0.0;
	var_t vx = 0.0;
	trans_to_descartes_var(x, vx);

	for (uint32_t i = 0; i < n_obj; i++)
    {
		sout << setw(VAR_T_W) << t << SEP                       /* 1  independent variable     (double)           */
		// Print the metadata for each object
        << setw(INT_T_W) << h_md[i].id << SEP;                  /* 2  id of the body           (int32_t)          */ 

		// Print the parameters for each object
		for (uint16_t j = 0; j < n_ppo; j++)
		{
			uint32_t param_idx = i * n_ppo + j;
			sout << setw(VAR_T_W) << h_p[param_idx] << SEP;     /* 3 mu of the problem        (double)            */
		}
		// Print the regularized variables for each object
		for (uint16_t j = 0; j < n_vpo; j++)                    /* 4 u (reg. coordinate) of the object  (double)  */
		{                                                       /* 5 v (reg. velocity) of the object    (double)  */
			uint32_t var_idx = i * n_vpo + j;                   /* 6 s (real time of the problem)       (double)  */
			sout << setw(VAR_T_W) << h_y[var_idx] << SEP;
		}
		// Print the descartes non-regularized variables for each object
		sout << setw(VAR_T_W) << x << SEP                       /* 7 x coordinate of the object         (double)  */
			 << setw(VAR_T_W) << vx << endl;                    /* 8 vx velocity  of the object         (double)  */
	}
	sout.flush();
}

void rtbp1D::print_solution_binary(ofstream& sout)
{
	throw string("The print_solution_binary() is not implemented.");
}

void rtbp1D::print_integral(string& path)
{
	ofstream sout;

	sout.open(path.c_str(), ios::out | ios::app);
	if (sout)
	{
		sout.precision(16);
		sout.setf(ios::right);
		sout.setf(ios::scientific);

		sout << setw(VAR_T_W) << t << SEP             /* fictitious time of the record (double)   */
			 << setw(VAR_T_W) << h_y[2] << SEP        /* real time of the record [day] double     */
			 << setw(VAR_T_W) << integral.h << endl;  /* energy of the system                     */
	}
	else
	{
		throw string("Cannot open " + path + ".");
	}
	sout.close();
}

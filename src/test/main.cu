#include "hip/hip_runtime.h"
/*
 * 2016.11.11. - 11.13. TEST OK
 * Allocation of array of pointers
 * Allocation of each element in the array
 */
#if 0
#include <stdio.h>      /* printf, scanf, puts, NULL */
#include <stdlib.h>     /* srand, rand, malloc       */
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "type.h"
#include "macro.h"
#include "redutil2.h"

using namespace redutil2;

namespace kernel_test
{
__global__
	void print_array(int **a, uint32_t n_vct)
	{
		const int tid = threadIdx.x;

		if (0 == tid)
		{
			for (uint32_t i = 0; i < n_vct; i++)
			{
				printf("[%u]: %p *(_+%u): %p\n", i, a[i], i, *(a+i));
			}
		}
	}

__global__
	void print_array(int *a, uint32_t n_arr)
	{
		const int tid = threadIdx.x;

		if (0 == tid)
		{
			for (uint32_t i = 0; i < n_arr; i++)
			{
				printf("\t[%u]: %d\n", i, a[i]);
			}
		}
	}
} /* kernel_test */

void print_array(int **a, uint32_t n_vct)
{
	for (uint32_t i = 0; i < n_vct; i++)
	{
		printf(" +%u: %p\t", i, a+i);
		printf("[%u]: %p *( +%u): %p\n", i, a[i], i, *(a+i));
	}
}

void print_array(int *a, uint32_t n_arr)
{
	for (uint32_t i = 0; i < n_arr; i++)
	{
		printf("\t[%u]: %d\n", i, a[i]);
	}
}

int main()
{
	static const uint32_t n_vct = 5;
	static const uint32_t n_arr = 9;

	int** h_k = NULL;
	int** d_k = NULL;
	int** tmp = NULL;

	try
	{
		printf("h_k: %p\t", h_k);

		// Allocate HOST memory
		ALLOCATE_HOST_VECTOR((void**)&h_k, n_vct*sizeof(int*));
		printf("after allocation: %p\n", h_k);

		for (uint32_t i = 0; i < n_vct; i++)
		{
			printf("h_k[%u]: %p\t", i, h_k[i]);
			ALLOCATE_HOST_VECTOR((void**)(h_k + i), n_arr*sizeof(int));
			printf("after allocation: %p\n", h_k[i]);
			print_array(*(h_k + i), n_arr);
		}

		printf("tmp: %p\t", tmp);
		ALLOCATE_HOST_VECTOR((void**)&tmp, n_vct*sizeof(int*));
		printf("after allocation: %p\n", tmp);

		// Allocate DEVICE memory
		printf("d_k: %p\t", d_k);
		ALLOCATE_DEVICE_VECTOR((void**)(&d_k), n_vct*sizeof(int*));
		printf("after allocation: %p\n", d_k);

		for (uint32_t i = 0; i < n_vct; i++)
		{
			printf("tmp[%u]: %p\t", i, tmp[i]);
			ALLOCATE_DEVICE_VECTOR((void**)(tmp + i), n_arr*sizeof(int));
			printf("after allocation: %p\n", tmp[i]);
			kernel_test::print_array<<<1,  1>>>(*(tmp + i), n_arr);
			hipDeviceSynchronize();
		}
		CUDA_SAFE_CALL(hipMemcpy(d_k, tmp, n_vct * sizeof(int*), hipMemcpyHostToDevice));
		kernel_test::print_array<<<1,  1>>>(d_k, n_vct);
		hipDeviceSynchronize();


		// Populate data
		for (uint32_t i = 0; i < n_vct; i++)
		{
			for (uint32_t j = 0; j < n_arr; j++)
			{
				*(*(h_k+i)+j) = i*10 + j;
			}
			printf("h_k[%u]: %p\n", i, h_k[i]);
			print_array(*(h_k + i), n_arr);
			printf("\n");

			printf("tmp[%u]: %p\n", i, tmp[i]);
			CUDA_SAFE_CALL(hipMemcpy(tmp[i], h_k[i], n_arr * sizeof(int), hipMemcpyHostToDevice));
			kernel_test::print_array<<<1,  1>>>(tmp[i], n_arr);
			hipDeviceSynchronize();
		}

		// Deallocate memory
		for (uint32_t i = 0; i < n_vct; i++)
		{
			FREE_HOST_VECTOR((void**)(h_k + i));
			FREE_DEVICE_VECTOR((void**)(tmp + i));
		}
		FREE_HOST_VECTOR((void**)&h_k);
		FREE_HOST_VECTOR((void**)&tmp);
		FREE_DEVICE_VECTOR((void**)&d_k);
	}
	catch (const std::string& msg)
	{
		std::cerr << "Error: " << msg << std::endl;
	}

	return 0;
}
#endif


/*
 * 2016.11.13. - 11.13.  TEST OK
 * Compute the linear combination of arrays on the DEVICE
 * and comapre the results those computed on the HOST
 */
#if 1
#include <stdio.h>      /* printf, scanf, puts, NULL */
#include <stdlib.h>     /* srand, rand, malloc       */
#include <time.h>       /* time                      */
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "type.h"
#include "macro.h"
#include "redutil2.h"

using namespace redutil2;

namespace kernel_test
{
__global__
void print_array(var_t *a, uint32_t n_arr)
{
	const int tid = threadIdx.x;

	if (0 == tid)
	{
		for (uint32_t i = 0; i < n_arr; i++)
		{
			printf("\t[%u]: %g\n", i, a[i]);
		}
	}
}

//! Calculate the special case of linear combination of vectors, a[i] = b[i] + sum (coeff[j] * c[j][i])
/*
	\param a     vector which will contain the result
	\param b     vector to which the linear combination will be added
	\param c     vectors which will linear combined
	\param coeff vector which contains the weights (coefficients)
	\param n_vct the number of vectors to combine
	\param n_var the number of elements in the vectors
*/
__global__
void calc_lin_comb_s(var_t* a, const var_t* b, const var_t* const *c, const var_t* coeff, uint16_t n_vct, uint32_t n_var)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < n_var)
	{
		var_t d = 0.0;
		for (uint16_t j = 0; j < n_vct; j++)
		{
			if (0.0 == coeff[j])
			{
				continue;
			}
			d += coeff[j] * c[j][tid];
		}
		a[tid] = b[tid] + d;
	}
}
} /* kernel_test */

int main()
{
	static const uint32_t n_vct = 5;
	static const uint32_t n_arr = 3000;

	var_t** h_k = NULL;
	var_t** d_k = NULL;
	var_t** tmp = NULL;

	var_t* h_a = NULL;
	var_t* h_a0 = NULL;     // Will hold a copy of d_a
	var_t* h_b = NULL;
	var_t* h_coeff = NULL;

	var_t* d_a = NULL;
	var_t* d_b = NULL;
	var_t* d_coeff = NULL;

	try
	{
		// Allocate HOST memory
		ALLOCATE_HOST_VECTOR((void**)&h_k, n_vct*sizeof(var_t*));
		for (uint32_t i = 0; i < n_vct; i++)
		{
			ALLOCATE_HOST_VECTOR((void**)(h_k + i), n_arr*sizeof(var_t));
		}
		ALLOCATE_HOST_VECTOR((void**)&tmp, n_vct*sizeof(var_t*));

		ALLOCATE_HOST_VECTOR((void**)&h_a,     n_arr*sizeof(var_t));
		ALLOCATE_HOST_VECTOR((void**)&h_a0,    n_arr*sizeof(var_t));
		ALLOCATE_HOST_VECTOR((void**)&h_b,     n_arr*sizeof(var_t));
		ALLOCATE_HOST_VECTOR((void**)&h_coeff, n_vct*sizeof(var_t));

		// Allocate DEVICE memory
		ALLOCATE_DEVICE_VECTOR((void**)(&d_k), n_vct*sizeof(var_t*));
		for (uint32_t i = 0; i < n_vct; i++)
		{
			ALLOCATE_DEVICE_VECTOR((void**)(tmp + i), n_arr*sizeof(var_t));
		}
		CUDA_SAFE_CALL(hipMemcpy(d_k, tmp, n_vct * sizeof(var_t*), hipMemcpyHostToDevice));

		ALLOCATE_DEVICE_VECTOR((void**)&d_a,     n_arr*sizeof(var_t));
		ALLOCATE_DEVICE_VECTOR((void**)&d_b,     n_arr*sizeof(var_t));
		ALLOCATE_DEVICE_VECTOR((void**)&d_coeff, n_vct*sizeof(var_t));

		// Populate data
		srand(time(NULL));
		for (uint32_t i = 0; i < n_vct; i++)
		{
			for (uint32_t j = 0; j < n_arr; j++)
			{
				var_t r = (var_t)rand()/RAND_MAX;    //returns a pseudo-random integer between 0 and RAND_MAX			
				*(*(h_k+i)+j) = r;
			}
			CUDA_SAFE_CALL(hipMemcpy(tmp[i], h_k[i], n_arr * sizeof(var_t), hipMemcpyHostToDevice));
		}
		for (uint32_t j = 0; j < n_arr; j++)
		{
			h_a[j] = 0;
			h_b[j] = 0;
		}
		for (uint32_t j = 0; j < n_vct; j++)
		{
			h_coeff[j] = 1;
		}
		h_coeff[4] = -1;

		CUDA_SAFE_CALL(hipMemcpy(d_a, h_a, n_arr * sizeof(var_t), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_b, h_b, n_arr * sizeof(var_t), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_coeff, h_coeff, n_vct * sizeof(var_t), hipMemcpyHostToDevice));

		// Test the tools::calc_lin_comb_s() and gpu_calc_lin_comb_s() functions
		// Compute a[i] = b[i] + f*c[i]
		{
			printf("Compute a[i] = b[i] + f*c[i]\n\n");
			var_t f = 2.0;
			var_t *h_c = *h_k;
			tools::calc_lin_comb_s(h_a, h_b, h_c, f, n_arr);

			var_t *d_c = *tmp;
			gpu_calc_lin_comb_s(   d_a, d_b, d_c, f, n_arr, 0, false);

			//printf("h_a:\n");
			//print_array("", n_arr, h_a, PROC_UNIT_CPU);

			//printf("d_a:\n");
			//print_array("", n_arr, d_a, PROC_UNIT_GPU);

			CUDA_SAFE_CALL(hipMemcpy(h_a0, d_a, n_arr * sizeof(var_t), hipMemcpyDeviceToHost));

			for (uint32_t j = 0; j < n_arr; j++)
			{
				if (0 != fabs(h_a[j] - h_a0[j]))
				{
					printf("Difference: j = %6u : %g\n", j, h_a[j] - h_a0[j]);
				}
			}
		}

		// Test the tools::calc_lin_comb_s() and gpu_calc_lin_comb_s() functions
		// Compute a[i] = b[i] + sum (coeff[j] * c[j][i])
		{
			printf("Compute a[i] = b[i] + sum (coeff[j] * c[j][i])\n\n");
			tools::calc_lin_comb_s(h_a, h_b, h_k, h_coeff, n_vct, n_arr);
			gpu_calc_lin_comb_s(   d_a, d_b, d_k, d_coeff, n_vct, n_arr, 0, false);
	
			CUDA_SAFE_CALL(hipMemcpy(h_a0, d_a, n_arr * sizeof(var_t), hipMemcpyDeviceToHost));

			for (uint32_t j = 0; j < n_arr; j++)
			{
				if (0 != fabs(h_a[j] - h_a0[j]))
				{
					printf("Difference: j = %6u : %g\n", j, h_a[j] - h_a0[j]);
				}
			}
		}

		// Deallocate memory
		for (uint32_t i = 0; i < n_vct; i++)
		{
			FREE_HOST_VECTOR((void**)(h_k + i));
			FREE_DEVICE_VECTOR((void**)(tmp + i));
		}
		FREE_HOST_VECTOR((void**)&h_k);
		FREE_HOST_VECTOR((void**)&tmp);
		FREE_DEVICE_VECTOR((void**)&d_k);

		FREE_HOST_VECTOR((void**)&h_a);
		FREE_HOST_VECTOR((void**)&h_a0);
		FREE_HOST_VECTOR((void**)&h_b);
		FREE_HOST_VECTOR((void**)&h_coeff);

		FREE_DEVICE_VECTOR((void**)&d_a);
		FREE_DEVICE_VECTOR((void**)&d_b);
		FREE_DEVICE_VECTOR((void**)&d_coeff);
	}
	catch (const std::string& msg)
	{
		std::cerr << "Error: " << msg << std::endl;
	}

	std::cout << "Compute the linear combination of arrays on the DEVICE and comapre the results those computed on the HOST done.\n";

	return 0;
}
#endif


/*
 * 2016.11.14. - 
 * Gravitational interaction computations
 */
#if 0
/*
Premature optimization is the ROOT OF ALL EVIL. Always remember the three rules of optimization!

1. Don't optimize.
2. If you are an expert, see rule #1
3. If you are an expert and can justify the need, then use the following procedure:
 - Code it unoptimized
 - determine how fast is "Fast enough"--Note which user requirement/story requires that metric.
 - Write a speed test
 - Test existing code--If it's fast enough, you're done.
 - Recode it optimized
 - Test optimized code. IF it doesn't meet the metric, throw it away and keep the original.
 - If it meets the test, keep the original code in as comments
*/

#include <stdio.h>      /* printf, scanf, puts, NULL */
#include <stdlib.h>     /* srand, rand, malloc       */
#include <time.h>       /* time                      */
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "type.h"
#include "macro.h"
#include "redutil2.h"

#ifdef _WIN32
#include <Windows.h>
#else
#include <sys/time.h>
#include <ctime>
#endif

using namespace redutil2;

// Global variables
uint32_t n_tpb = 128;

uint32_t n_obj = 0;
var_t* h_p = NULL;
var_t* d_p = NULL;

dim3 grid;
dim3 block;


namespace nbody_kernel
{
__global__
void calc_gravity_accel_naive(uint32_t n_obj, const var3_t* r, const nbp_t::param_t* p, var3_t* a)
{
	const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (n_obj > i)
	{
		var3_t r_ij = {0, 0, 0};
		for (uint32_t j = 0; j < n_obj; j++)
		{
			if (i == j)
			{
				continue;
			}
			r_ij.x = r[j].x - r[i].x;
			r_ij.y = r[j].y - r[i].y;
			r_ij.z = r[j].z - r[i].z;

			var_t d2 = SQR(r_ij.x) + SQR(r_ij.y) + SQR(r_ij.z);
			var_t d = sqrt(d2);
			var_t d_3 = 1.0 / (d*d2);

			var_t s = p[j].mass * d_3;
			a[i].x += s * r_ij.x;
			a[i].y += s * r_ij.y;
			a[i].z += s * r_ij.z;
		}
		a[i].x *= K2;
		a[i].y *= K2;
		a[i].z *= K2;
	}
}

__global__
void calc_gravity_accel_naive_sym(uint32_t n_obj, const var3_t* r, const nbp_t::param_t* p, var3_t* a)
{
	const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (n_obj > i)
	{
		var3_t r_ij = {0, 0, 0};
		for (uint32_t j = i+1; j < n_obj; j++)
		{
			r_ij.x = r[j].x - r[i].x;
			r_ij.y = r[j].y - r[i].y;
			r_ij.z = r[j].z - r[i].z;

			var_t d2 = SQR(r_ij.x) + SQR(r_ij.y) + SQR(r_ij.z);
			var_t d = sqrt(d2);
			var_t d_3 = 1.0 / (d*d2);

			var_t s = p[j].mass * d_3;
			a[i].x += s * r_ij.x;
			a[i].y += s * r_ij.y;
			a[i].z += s * r_ij.z;

			s = p[i].mass * d_3;
			a[j].x -= s * r_ij.x;
			a[j].y -= s * r_ij.y;
			a[j].z -= s * r_ij.z;
		}
		a[i].x *= K2;
		a[i].y *= K2;
		a[i].z *= K2;
	}
}

inline __host__ __device__
	var3_t body_body_interaction(var3_t riVec, var3_t rjVec, var_t mj, var3_t aiVec)
{
	var3_t dVec = {0.0, 0.0, 0.0};

	// compute d = r_i - r_j [3 FLOPS] [6 read, 3 write]
	dVec.x = rjVec.x - riVec.x;
	dVec.y = rjVec.y - riVec.y;
	dVec.z = rjVec.z - riVec.z;

	// compute norm square of d vector [5 FLOPS] [3 read, 1 write]
	var_t r2 = SQR(dVec.x) + SQR(dVec.y) + SQR(dVec.z);
	// compute norm of d vector [1 FLOPS] [1 read, 1 write] TODO: how long does it take to compute sqrt ???
	var_t r = sqrt(r2);
	// compute m_j / d^3 []
	var_t s = mj * 1.0 / (r2 * r);

	aiVec.x += s * dVec.x;
	aiVec.y += s * dVec.y;
	aiVec.z += s * dVec.z;

	return aiVec;
}

__global__
	void calc_gravity_accel_tile(interaction_bound int_bound, int tile_size, const var3_t* r, const nbp_t::param_t* p, var3_t* a)
{
	extern __shared__ var3_t sh_pos[];

	var3_t my_pos = {0.0, 0.0, 0.0};
	var3_t acc    = {0.0, 0.0, 0.0};

	// i is the index of the SINK body
	const uint32_t i = int_bound.sink.x + blockIdx.x * blockDim.x + threadIdx.x;

	// To avoid overruning the r buffer
	if (int_bound.sink.y > i)
	{
		my_pos = r[i];
	}
	for (int tile = 0; (tile * tile_size) < int_bound.source.y; tile++)
	{
		// src_idx is the index of the SOURCE body in the tile
		int src_idx = int_bound.source.x + tile * tile_size + threadIdx.x;
		// To avoid overruning the r buffer
		if (int_bound.source.y > src_idx)
		{
			sh_pos[threadIdx.x] = r[src_idx];
		}
		__syncthreads();
		// j is the index of the SOURCE body in the current tile
		for (int j = 0; j < blockDim.x; j++)
		{
			// To avoid overrun the mass buffer
			if (int_bound.source.y <= int_bound.source.x + (tile * tile_size) + j)
			{
				break;
			}
			// To avoid self-interaction or mathematically division by zero
			if (i != int_bound.source.x + (tile * tile_size)+j)
			{
				acc = body_body_interaction(my_pos, sh_pos[j], p[src_idx].mass, acc);
			}
		}
		__syncthreads();
	}

	// To avoid overruning the a buffer
	if (int_bound.sink.y > i)
	{
		a[i] = acc;
	}
}
} /* nbody_kernel */


/* 
 *  -- Returns the amount of milliseconds elapsed since the UNIX epoch. Works on both --
 * Returns the amount of microseconds elapsed since the UNIX epoch. Works on both
 * windows and linux.
 */
uint64_t GetTimeMs64()
{
#ifdef _WIN32
	/* Windows */
	FILETIME ft;
	LARGE_INTEGER li;

	/* Get the amount of 100 nano seconds intervals elapsed since January 1, 1601 (UTC) and copy it
	* to a LARGE_INTEGER structure. */
	GetSystemTimeAsFileTime(&ft);
	li.LowPart = ft.dwLowDateTime;
	li.HighPart = ft.dwHighDateTime;

	uint64_t ret = li.QuadPart;
	ret -= 116444736000000000LL; /* Convert from file time to UNIX epoch time. */
	//ret /= 10000; /* From 100 nano seconds (10^-7) to 1 millisecond (10^-3) intervals */
	ret /= 10; /* From 100 nano seconds (10^-7) to 1 microsecond (10^-6) intervals */

	return ret;
#else
	/* Linux */
	struct timeval tv;

	gettimeofday(&tv, NULL);

	uint64 ret = tv.tv_usec;
	/* Convert from micro seconds (10^-6) to milliseconds (10^-3) */
	//ret /= 1000;

	/* Adds the seconds (10^0) after converting them to milliseconds (10^-3) */
	//ret += (tv.tv_sec * 1000);
	/* Adds the seconds (10^0) after converting them to microseconds (10^-6) */
	ret += (tv.tv_sec * 1000000);

	return ret;
#endif
}


float gpu_calc_dy(uint32_t n_var, uint16_t stage, var_t curr_t, const var_t* y_temp, var_t* dy, bool use_symm_prop)
{
	set_kernel_launch_param(n_var, n_tpb, grid, block);
		
	printf(" grid: (%4u, %4u, %4u)\n", grid.x, grid.y, grid.z);
	printf("block: (%4u, %4u, %4u)\n", block.x, block.y, block.z);

	var3_t* r = (var3_t*)y_temp;
	var3_t* a = (var3_t*)(dy + 3*n_obj);
	nbp_t::param_t* p = (nbp_t::param_t*)d_p;

	hipEvent_t t0, t1;
	CUDA_SAFE_CALL(hipEventCreate(&t0));
	CUDA_SAFE_CALL(hipEventCreate(&t1));

	CUDA_SAFE_CALL(hipEventRecord(t0));
	// Clear the acceleration array: the += op can be used
	CUDA_SAFE_CALL(hipMemset(a, 0, n_obj*sizeof(var3_t)));

	// Copy the velocities into dy
	// TODO: implement the asynchronous version of hipMemcpy: Performace ??
	CUDA_SAFE_CALL(hipMemcpy(dy, y_temp + 3*n_obj, 3*n_obj*sizeof(var_t), hipMemcpyDeviceToDevice));

	if (false == use_symm_prop)
	{
		nbody_kernel::calc_gravity_accel_naive<<<grid, block>>>(n_obj, r, p, a);
	}
	else
	{
		nbody_kernel::calc_gravity_accel_naive_sym<<<grid, block>>>(n_obj, r, p, a);
	}
	CUDA_CHECK_ERROR();
	CUDA_SAFE_CALL(hipEventRecord(t1));
	CUDA_SAFE_CALL(hipEventSynchronize(t1));

	float dt = 0.0f;
	CUDA_SAFE_CALL(hipEventElapsedTime(&dt, t0, t1));

	return dt;
}

float gpu_calc_grav_accel_tile(uint32_t n_var, uint16_t stage, var_t curr_t, const var_t* y_temp, var_t* dy)
{
	set_kernel_launch_param(n_var, n_tpb, grid, block);
		
	printf(" grid: (%4u, %4u, %4u)\n", grid.x, grid.y, grid.z);
	printf("block: (%4u, %4u, %4u)\n", block.x, block.y, block.z);

	uint2_t sink   = {0, n_obj};
	uint2_t source = {0, n_obj};
	interaction_bound int_bound(sink, source);

	var3_t* r = (var3_t*)y_temp;
	var3_t* a = (var3_t*)(dy + 3*n_obj);
	nbp_t::param_t* p = (nbp_t::param_t*)d_p;

	hipEvent_t t0, t1;
	CUDA_SAFE_CALL(hipEventCreate(&t0));
	CUDA_SAFE_CALL(hipEventCreate(&t1));

	CUDA_SAFE_CALL(hipEventRecord(t0));
	// Clear the acceleration array: the += op can be used
	CUDA_SAFE_CALL(hipMemset(a, 0, n_obj*sizeof(var3_t)));

	// Copy the velocities into dy
	// TODO: implement the asynchronous version of hipMemcpy: Performace ??
	CUDA_SAFE_CALL(hipMemcpy(dy, y_temp + 3*n_obj, 3*n_obj*sizeof(var_t), hipMemcpyDeviceToDevice));

	nbody_kernel::calc_gravity_accel_tile<<<grid, block, n_tpb * sizeof(var3_t)>>>(int_bound, n_tpb, r, p, a);
	CUDA_CHECK_ERROR();

	CUDA_SAFE_CALL(hipEventRecord(t1, 0));
	CUDA_SAFE_CALL(hipEventSynchronize(t1));

	float elapsed_time = 0.0f;
	CUDA_SAFE_CALL(hipEventElapsedTime(&elapsed_time, t0, t1));

	return elapsed_time;
}

void cpu_calc_dy(uint16_t stage, var_t curr_t, const var_t* y_temp, var_t* dy, bool use_symm_prop)
{
	// Copy the velocities into dy
	memcpy(dy, y_temp + 3*n_obj, 3*n_obj*sizeof(var_t));

	var3_t* r = (var3_t*)y_temp;
	var3_t* a = (var3_t*)(dy + 3*n_obj);
	// Clear the acceleration array: the += op can be used
	memset(a, 0, 3*n_obj*sizeof(var_t));

	nbp_t::param_t* p = (nbp_t::param_t*)h_p;

	if (use_symm_prop)
	{
		for (uint32_t i = 0; i < n_obj; i++)
		{
			var3_t r_ij = {0, 0, 0};
			for (uint32_t j = i+1; j < n_obj; j++)
			{
				r_ij.x = r[j].x - r[i].x;
				r_ij.y = r[j].y - r[i].y;
				r_ij.z = r[j].z - r[i].z;

				var_t d2 = SQR(r_ij.x) + SQR(r_ij.y) + SQR(r_ij.z);
				var_t d = sqrt(d2);
				var_t d_3 = 1.0 / (d*d2);

				var_t s = p[j].mass * d_3;
				a[i].x += s * r_ij.x;
				a[i].y += s * r_ij.y;
				a[i].z += s * r_ij.z;

				s = p[i].mass * d_3;
				a[j].x -= s * r_ij.x;
				a[j].y -= s * r_ij.y;
				a[j].z -= s * r_ij.z;
			}
			a[i].x *= K2;
			a[i].y *= K2;
			a[i].z *= K2;
		}
	}
	else
	{
		for (uint32_t i = 0; i < n_obj; i++)
		{
			var3_t r_ij = {0, 0, 0};
			for (uint32_t j = 0; j < n_obj; j++)
			{
				if (i == j)
				{
					continue;
				}
				r_ij.x = r[j].x - r[i].x;
				r_ij.y = r[j].y - r[i].y;
				r_ij.z = r[j].z - r[i].z;

				var_t d2 = SQR(r_ij.x) + SQR(r_ij.y) + SQR(r_ij.z);
				var_t d = sqrt(d2);
				var_t d_3 = 1.0 / (d*d2);

				var_t s = p[j].mass * d_3;
				a[i].x += s * r_ij.x;
				a[i].y += s * r_ij.y;
				a[i].z += s * r_ij.z;
			}
			a[i].x *= K2;
			a[i].y *= K2;
			a[i].z *= K2;
		}
	}
}

void parse(int argc, const char** argv, uint32_t* n_obj)
{
	int i = 1;

	if (1 >= argc)
	{
		throw std::string("Missing command line arguments. For help use -h.");
	}

	while (i < argc)
	{
		std::string p = argv[i];
		if (     p == "-n")
		{
			i++;
			if (!tools::is_number(argv[i])) 
			{
				throw std::string("Invalid number at: " + p);
			}
			*n_obj = atoi(argv[i]);
		}
		else
		{
			throw std::string("Invalid switch on command line: " + p + ".");
		}
		i++;
	}
}

int main(int argc, const char *argv[])
{
	var_t* h_y = NULL;
	var_t* h_dy = NULL;
	var_t* h_dy0 = NULL;

	var_t* d_y = NULL;
	var_t* d_dy = NULL;

	uint32_t n_var = 0;
	uint32_t n_par = 0;

	try
	{
		// n_obj is a global variable
		parse(argc, argv, &n_obj);
		n_var = 6 * n_obj;
		n_par = 1 * n_obj;

		// Allocate HOST memory
		ALLOCATE_HOST_VECTOR((void**)&h_y,   n_var * sizeof(var_t));
		ALLOCATE_HOST_VECTOR((void**)&h_dy,  n_var * sizeof(var_t));
		ALLOCATE_HOST_VECTOR((void**)&h_dy0, n_var * sizeof(var_t));
		ALLOCATE_HOST_VECTOR((void**)&h_p,   n_par * sizeof(var_t));

		// Allocate DEVICE memory
		ALLOCATE_DEVICE_VECTOR((void**)&d_y,  n_var * sizeof(var_t));
		ALLOCATE_DEVICE_VECTOR((void**)&d_dy, n_var * sizeof(var_t));
		ALLOCATE_DEVICE_VECTOR((void**)&d_p,  n_par * sizeof(var_t));

		// Populate data
		srand(time(NULL));
		for (uint32_t i = 0; i < n_var; i++)
		{
			var_t r = (var_t)rand()/RAND_MAX;
			*(h_y + i) = r;
		}
		for (uint32_t i = 0; i < n_par; i++)
		{
			var_t r = (var_t)rand()/RAND_MAX;
			*(h_p + i) = 1;
		}

		CUDA_SAFE_CALL(hipMemcpy(d_y, h_y, n_var * sizeof(var_t), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_p, h_p, n_par * sizeof(var_t), hipMemcpyHostToDevice));

		var_t t0 = 0.0;
		uint16_t stage = 0;

		uint64_t T0 = GetTimeMs64();
		cpu_calc_dy(stage, t0, h_y, h_dy, false);
		uint64_t T1 = GetTimeMs64();
		var_t DT_CPU = ((var_t)(T1 - T0))/1000.0f;
		printf("CPU execution time: %16.4e [ms]\n", DT_CPU);

		T0 = GetTimeMs64();
		cpu_calc_dy(stage, t0, h_y, h_dy0, true);
		T1 = GetTimeMs64();
		DT_CPU = ((var_t)(T1 - T0))/1000.0f;
		printf("CPU execution time: %16.4e [ms]\n", DT_CPU);

		for (uint32_t j = 0; j < n_var; j++)
		{
			if (1.0e-15 < fabs(h_dy[j] - h_dy0[j]))
			{
				printf("Difference: j = %6u : %24.16e\n", j, h_dy[j] - h_dy0[j]);
			}
		}

		T0 = GetTimeMs64();
		float _DT_GPU = gpu_calc_dy(n_var, stage, t0, d_y, d_dy, false);
		T1 = GetTimeMs64();
		var_t DT_GPU = ((var_t)(T1 - T0))/1000.0f;
		printf("GPU execution time: %16.4e [ms]\n", DT_GPU);
		printf("GPU execution time: %16.4e [ms]\n", _DT_GPU);
		printf("%10u %16.4e %16.4e %16.4e %16.4e\n", n_obj, DT_CPU, DT_GPU, _DT_GPU, DT_CPU/_DT_GPU);

		// Copy down the data from the DEVICE
		CUDA_SAFE_CALL(hipMemcpy(h_dy0, d_dy, n_var * sizeof(var_t), hipMemcpyDeviceToHost));

		for (uint32_t j = 0; j < n_var; j++)
		{
			if (1.0e-15 < fabs(h_dy[j] - h_dy0[j]))
			{
				printf("Difference: j = %6u : %24.16e\n", j, h_dy[j] - h_dy0[j]);
			}
		}

		T0 = GetTimeMs64();
		_DT_GPU = gpu_calc_dy(n_var, stage, t0, d_y, d_dy, true);
		T1 = GetTimeMs64();
		DT_GPU = ((var_t)(T1 - T0))/1000.0f;
		printf("GPU execution time: %16.4e [ms]\n", DT_GPU);
		printf("GPU execution time: %16.4e [ms]\n", _DT_GPU);
		printf("%10u %16.4e %16.4e %16.4e %16.4e\n", n_obj, DT_CPU, DT_GPU, _DT_GPU, DT_CPU/_DT_GPU);

		// Copy down the data from the DEVICE
		CUDA_SAFE_CALL(hipMemcpy(h_dy0, d_dy, n_var * sizeof(var_t), hipMemcpyDeviceToHost));

		for (uint32_t j = 0; j < n_var; j++)
		{
			if (1.0e-15 < fabs(h_dy[j] - h_dy0[j]))
			{
				printf("Difference: j = %6u : %24.16e\n", j, h_dy[j] - h_dy0[j]);
			}
		}

		T0 = GetTimeMs64();
		_DT_GPU = gpu_calc_grav_accel_tile(n_var, stage, t0, d_y, d_dy);
		T1 = GetTimeMs64();
		DT_GPU = ((var_t)(T1 - T0))/1000.0f;
		printf("GPU execution time: %16.4e [ms]\n", DT_GPU);
		printf("GPU execution time: %16.4e [ms]\n", _DT_GPU);
		printf("%10u %16.4e %16.4e %16.4e %16.4e\n", n_obj, DT_CPU, DT_GPU, _DT_GPU, DT_CPU/_DT_GPU);

		// Copy down the data from the DEVICE
		CUDA_SAFE_CALL(hipMemcpy(h_dy0, d_dy, n_var * sizeof(var_t), hipMemcpyDeviceToHost));

		for (uint32_t j = 0; j < n_var; j++)
		{
			if (1.0e-15 < fabs(h_dy[j] - h_dy0[j]))
			{
				printf("Difference: j = %6u : %24.16e\n", j, h_dy[j] - h_dy0[j]);
			}
		}

		FREE_HOST_VECTOR((void**)&h_y  );
		FREE_HOST_VECTOR((void**)&h_dy );
		FREE_HOST_VECTOR((void**)&h_dy0);
		FREE_HOST_VECTOR((void**)&h_p  );

		FREE_DEVICE_VECTOR((void**)&d_y );
		FREE_DEVICE_VECTOR((void**)&d_dy);
		FREE_DEVICE_VECTOR((void**)&d_p );
	}
	catch (const std::string& msg)
	{
		std::cerr << "Error: " << msg << std::endl;
	}
	std::cout << "Gravitational interaction computations done.\n";

	return 0;
}

#endif

#if 0
#include <stdio.h>      /* printf, scanf, puts, NULL */
#include <stdlib.h>     /* srand, rand, malloc       */
#include <time.h>       /* time                      */

#include <iostream>
#include <string>

#include "constants.h"
#include "type.h"
#include "redutil2.h"

using namespace std;
using namespace redutil2;

int comp_value(var_t v1, var_t v2, var_t tol, char* lpad, char* text)
{
	int result = 0;

	var_t d = fabs(v1 - v2);
	if (tol < d)
	{
		printf("%s%s = %25.15lg\n", lpad, text, d);
		result = 1;
	}

	return result;
}

int comp_oe(orbelem_t &oe1, orbelem_t& oe2, var_t tol, char* lpad)
{
	int result = comp_value(oe1.sma, oe2.sma, tol, lpad, "Abs(Delta(sma ))");
	result += comp_value(oe1.ecc, oe2.ecc, tol, lpad, "Abs(Delta(ecc ))");
	result += comp_value(oe1.inc, oe2.inc, tol, lpad, "Abs(Delta(inc ))");
	result += comp_value(oe1.peri, oe2.peri, tol, lpad, "Abs(Delta(peri))");
	result += comp_value(oe1.node, oe2.node, tol, lpad, "Abs(Delta(node))");
	result += comp_value(oe1.mean, oe2.mean, tol, lpad, "Abs(Delta(mean))");
	return result;
}

int comp_2D_vectors(var2_t &v1, var2_t &v2, var_t tol, char* lpad)
{
	int result = comp_value(v1.x, v2.x, tol, lpad, "Abs(Delta(v1.x - v2.x))");
	result += comp_value(v1.y, v2.y, tol, lpad, "Abs(Delta(v1.y - v2.y))");
	return result;
}

var_t random(var_t x0, var_t x1)
{
	return (x0 + ((var_t)rand() / RAND_MAX) * (x1 - x0));
}

void test_calc_ephemeris()
{
	// Test calculate phase from orbital elements and vice versa
	{
		const char func_name[] = "calc_phase";
		char lpad[] = "        ";
		/*
		 * The units are:
		 *     Unit name         | Unit symbol | Quantity name
		 *     -----------------------------------------------
		 *     Astronomical unit |          AU | length
		 *     Solar mass        |           S | mass
		 *     Mean solar day    |           D | time
		 */

		srand((unsigned int)time(NULL));
		// parameter of the problem
		tbp_t::param_t p;
            
		// Set the parameter of the problem
		p.mu = constants::Gauss2 * (1.0 + 1.0);
		// Set the initial orbital elements
		orbelem_t oe1 = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
		orbelem_t oe2 = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
		var3_t r0 = {0, 0, 0};
		var3_t v0 = {0, 0, 0};

		var_t tol = 1.0e-14;
		for (int i = 0; i < 100; i++)
		{
			oe1.sma = random(0.1, 10.0);
			oe1.ecc = random(0.0, 0.8);
			oe1.inc = random(0.0, PI);
			oe1.peri =random(0.0, TWOPI);
			oe1.node =random(0.0, TWOPI);
			oe1.mean =random(0.0, TWOPI);
			// Calculate the position and velocity vectors from orbital elements
			tools::calc_phase(p.mu, &oe1, &r0, &v0);
			// Calculate the orbital elements from position and velocity vectors
			tools::calc_oe(p.mu, &r0, &v0, &oe2);
	
			int ret_val = comp_oe(oe1, oe2, tol, lpad);
			if (0 < ret_val)
			{
				printf("    TEST '%s' failed with tolerance level: %g\n", func_name, tol);
			}
			else
			{
				printf("    TEST '%s' passed with tolerance level: %g\n", func_name, tol);
			}
		}
	} /* Test calc_phase() and calc_oe() functions */
}

void test_rtbp2d_calc_energy()
{
	// Test tools::tbp::calc_integral() and tools::rtbp2D::calc_integral() functions
	{
		const char func_name[] = "tools::tbp::calc_integral";
		char lpad[] = "        ";

	    /*
	     * The units are:
	     *     Unit name         | Unit symbol | Quantity name
	     *     -----------------------------------------------
	     *     Astronomical unit |          AU | length
	     *     Solar mass        |           S | mass
	     *     Mean solar day    |           D | time
	     */

		srand(0);

		orbelem_t oe = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
		var3_t r0 = {0, 0, 0};
		var3_t v0 = {0, 0, 0};

		var_t mu = constants::Gauss2*(1.0 + 1.0);
		var_t tol = 1.0e-15;
		for (int i = 0; i < 10; i++)
		{
			// Set the initial orbital elements
			oe.sma  = random(0.1, 10.0);
			oe.ecc  = random(0.0, 0.8);
			oe.inc  = 0.0;
			oe.peri = random(0.0, TWOPI);
			oe.node = 0.0;
			oe.mean = random(0.0, TWOPI);
			// Calculate the position and velocity vectors from orbital elements
			tools::calc_phase(mu, &oe, &r0, &v0);

			// Set the starting coordinate and velocity vectors
			var2_t r  = {r0.x, r0.y};
			var2_t v  = {v0.x, v0.y};
			var2_t u  = {0, 0};
			var2_t up = {0, 0};
			tools::rtbp2D::transform_x2u(r, u);
			tools::rtbp2D::transform_xd2up(u, v, up);

			var_t hs = tools::tbp::calc_integral(mu, r, v);
			var_t hr = tools::rtbp2D::calc_integral(mu, u, up);

			printf("    hs = %25.15le\n", hs);
			printf("    hr = %25.15le\n", hr);
		}

		// Calculate the energy along a Kepler-orbit
		oe.sma  = 1.5;
		oe.ecc  = 0.8;
		oe.inc  = 0.0;
		oe.peri = 0.0;
		oe.node = 0.0;
		oe.mean = 0.0;
		do
		{
			tools::calc_phase(mu, &oe, &r0, &v0);
			var2_t r  = {r0.x, r0.y};
			var2_t v  = {v0.x, v0.y};
			var2_t u  = {0, 0};
			var2_t up = {0, 0};
			tools::rtbp2D::transform_x2u(r, u);
			tools::rtbp2D::transform_xd2up(u, v, up);

			var_t hs = tools::tbp::calc_integral(mu, r, v);
			var_t hr = tools::rtbp2D::calc_integral(mu, u, up);
			printf("%25.15le %25.15le %25.15le\n", oe.mean, hs, hr);

			oe.mean += 1.0 * constants::DegreeToRadian;
		} while (oe.mean <= TWOPI);
	} /* Test tools::rtbp2D::transform_x2u() and tools::rtbp2D::transform_u2x() functions */
}

void test_rtbp2d_transform()
{
	// Test square (section lines)
	{
		var_t d = 0.01;
		// Q4 -> Q1
		var2_t x = {0.5, -0.5};
		var2_t u  = {0, 0};
		do
		{
			tools::rtbp2D::transform_x2u(x, u);
			printf("%23.15le %23.15le %23.15le %23.15le\n", x.x, x.y, u.x, u.y);
			x.y += d;
		} while (0.5 >= x.y);
		// Q1 -> Q2
		do
		{
			tools::rtbp2D::transform_x2u(x, u);
			printf("%23.15le %23.15le %23.15le %23.15le\n", x.x, x.y, u.x, u.y);
			x.x -= d;
		} while (-0.5 <= x.x);
		// Q2 -> Q3
		do
		{
			tools::rtbp2D::transform_x2u(x, u);
			printf("%23.15le %23.15le %23.15le %23.15le\n", x.x, x.y, u.x, u.y);
			x.y -= d;
		} while (-0.5 <= x.y);
		// Q3 -> Q4
		do
		{
			tools::rtbp2D::transform_x2u(x, u);
			printf("%23.15le %23.15le %23.15le %23.15le\n", x.x, x.y, u.x, u.y);
			x.x += d;
		} while (0.5 >= x.x);
	}

	return;

	// Test ellipse
	{
		const char func_name[] = "tools::rtbp2D::transform___";
		char lpad[] = "        ";

	    /*
	     * The units are:
	     *     Unit name         | Unit symbol | Quantity name
	     *     -----------------------------------------------
	     *     Astronomical unit |          AU | length
	     *     Solar mass        |           S | mass
	     *     Mean solar day    |           D | time
	     */

		srand(0);

		const var_t mu = constants::Gauss2*(1.0 + 1.0);
		orbelem_t oe = {0.5, 0.8, 0.0, 0.0, 0.0, 0.0};
		var3_t r0 = {0, 0, 0};
		var3_t v0 = {0, 0, 0};
		int i = 0;
		do
		{
			oe.mean = i * constants::DegreeToRadian;
			tools::calc_phase(mu, &oe, &r0, &v0);
			var2_t x  = {r0.x, r0.y};
			var2_t xd = {v0.x, v0.y};
			var2_t u  = {0, 0};
			var2_t up = {0, 0};

			tools::rtbp2D::transform_x2u(x, u);
			tools::rtbp2D::transform_xd2up(u, xd, up);
			x.x  = x.y  = 0.0;
			xd.x = xd.y = 0.0;

			tools::rtbp2D::transform_u2x(u, x);
			tools::rtbp2D::transform_up2xd(u, up, xd);
			// Compare the original position and velocitiy vectors with the calculated ones
			{
				var_t tol = 1.0e-15;
				var2_t x0  = {r0.x, r0.y};
				var2_t x0d = {v0.x, v0.y};
				comp_2D_vectors(x0, x, tol, lpad);
				comp_2D_vectors(x0d, xd, tol, lpad);
			}

			printf("%23.15le %23.15le %23.15le %23.15le %23.15le %23.15le %23.15le %23.15le %23.15le\n", oe.mean, x.x, x.y, u.x, u.y, xd.x, xd.y, up.x, up.y);
			if (0 < i && 0 == (i+1) % 90)
			{
				printf("\n");
			}
			i++;
		} while (360 > i);
	} /* Test tools::rtbp2D::transform_x2u() and tools::rtbp2D::transform_u2x() functions */
}

void test_calc_lin_comb()
{
	// Test calculate linear combination of vectors
	{
		const char func_name[] = "calc_lin_comb";
		char lpad[] = "        ";

		uint32_t n_var = 2;
		uint16_t n_vct = 3;

		var_t* a = NULL;
		var_t* b = NULL;
		var_t** c = NULL;
		var_t* coeff = NULL;

		ALLOCATE_HOST_VECTOR((void**)&(a), n_var*sizeof(var_t));
		ALLOCATE_HOST_VECTOR((void**)&(b), n_var*sizeof(var_t));
		
		ALLOCATE_HOST_VECTOR((void**)&c, n_vct*sizeof(var_t*));
		for (uint16_t i = 0; i < n_vct; i++)
		{
			ALLOCATE_HOST_VECTOR((void**)&(c[i]), n_var*sizeof(var_t));
		}
		ALLOCATE_HOST_VECTOR((void**)&coeff, n_vct*sizeof(var_t));
	
		// Populate vectors
		memset(a, 0, n_var*sizeof(var_t));
		for (int i = 0; i < n_var; i++)
		{
			b[i] = -(i+1);
		}
		for (uint32_t i = 0; i < n_vct; i++)
		{
			for (uint32_t j = 0; j < n_var; j++)
			{
				c[i][j] = i+j+1;
			}
		}
		for (int i = 0; i < n_vct; i++)
		{
			coeff[i] = 10*i;
		}

		printf("The data in the vectors:\n");
		printf("a:\n");
		print_array("", n_var, a, PROC_UNIT_CPU);
		printf("b:\n");
		print_array("", n_var, b, PROC_UNIT_CPU);
		for (uint32_t i = 0; i < n_vct; i++)
		{
			printf("c[%d]:\n", i);
			print_array("", n_var, c[i], PROC_UNIT_CPU);
		}
		printf("The coefficients:\n");
		print_array("", n_vct, coeff, PROC_UNIT_CPU);

		// Calculate the linear combination of the vectors
		tools::calc_lin_comb(a, c, coeff, n_vct, n_var);
		printf("The linear combination of the vectors:\n");
		printf("a:\n");
		print_array("", n_var, a, PROC_UNIT_CPU);

		// Calculate the special case of linear combination of the vectors
		tools::calc_lin_comb_s(a, b, c, coeff, n_vct, n_var);
		printf("The special linear combination of the vectors:\n");
		printf("a:\n");
		print_array("", n_var, a, PROC_UNIT_CPU);

		FREE_HOST_VECTOR((void **)&(coeff));
		for (uint16_t i = 0; i < n_vct; i++)
		{
			FREE_HOST_VECTOR((void **)&(c[i]));
		}
		FREE_HOST_VECTOR((void **)&(c));
		FREE_HOST_VECTOR((void **)&(b));
		FREE_HOST_VECTOR((void **)&(a));
	}	

	// Test calculate linear combination of two vectors
	{
		const char func_name[] = "calc_lin_comb_s";
		char lpad[] = "        ";

		uint32_t n_var = 2;

		var_t* a = NULL;
		var_t* b = NULL;
		var_t* c = NULL;
		var_t f = 3;

		ALLOCATE_HOST_VECTOR((void**)&(a), n_var*sizeof(var_t));
		ALLOCATE_HOST_VECTOR((void**)&(b), n_var*sizeof(var_t));
		ALLOCATE_HOST_VECTOR((void**)&(c), n_var*sizeof(var_t));	

		// Populate vectors
		memset(a, 0, n_var*sizeof(var_t));
		for (int i = 0; i < n_var; i++)
		{
			b[i] = -(i+1);
			c[i] = i+1;
		}

		printf("The data in the vectors:\n");
		printf("a:\n");
		print_array("", n_var, a, PROC_UNIT_CPU);
		printf("b:\n");
		print_array("", n_var, b, PROC_UNIT_CPU);
		printf("c:\n");
		print_array("", n_var, c, PROC_UNIT_CPU);
		printf("The coefficient:\n");
		printf("%5e\n", f);

		// Calculate the special case of linear combination of the vectors
		tools::calc_lin_comb_s(a, b, c, f, n_var);
		printf("The special linear combination of two vectors:\n");
		printf("a:\n");
		print_array("", n_var, a, PROC_UNIT_CPU);

		FREE_HOST_VECTOR((void **)&(c));
		FREE_HOST_VECTOR((void **)&(b));
		FREE_HOST_VECTOR((void **)&(a));
	}	
}

/*

cd 'C:\Work\red.cuda.Results\v2.0\Test_Copy\rtbp2D\Test_transform
a=1.0
p [-1:1][-1:1]'e_0.0_q1.txt' u 2:3 w l, '' u 4:5 w l, 'e_0.0_q2.txt' u 2:3 w l, '' u 4:5 w l, 'e_0.0_q3.txt' u 2:3 w l, '' u 4:5 w l, 'e_0.0_q4.txt' u 2:3 w l, '' u 4:5 w l
a=0.05
p [-a:a][-a:a]'e_0.0_q1.txt' u 6:7 w l, '' u 8:9 w l, 'e_0.0_q2.txt' u 6:7 w l, '' u 8:9 w l, 'e_0.0_q3.txt' u 6:7 w l, '' u 8:9 w l, 'e_0.0_q4.txt' u 6:7 w l, '' u 8:9 w l

a=1.0
p [-1:1][-1:1]'e_0.2_q1.txt' u 2:3 w l, '' u 4:5 w l, 'e_0.2_q2.txt' u 2:3 w l, '' u 4:5 w l, 'e_0.2_q3.txt' u 2:3 w l, '' u 4:5 w l, 'e_0.2_q4.txt' u 2:3 w l, '' u 4:5 w l
a=0.05
p [-a:a][-a:a]'e_0.2_q1.txt' u 6:7 w l, '' u 8:9 w l, 'e_0.2_q2.txt' u 6:7 w l, '' u 8:9 w l, 'e_0.2_q3.txt' u 6:7 w l, '' u 8:9 w l, 'e_0.2_q4.txt' u 6:7 w l, '' u 8:9 w l

a=1.0
p [-1:1][-1:1]'e_0.8_q1.txt' u 2:3 w l, '' u 4:5 w l, 'e_0.8_q2.txt' u 2:3 w l, '' u 4:5 w l, 'e_0.8_q3.txt' u 2:3 w l, '' u 4:5 w l, 'e_0.8_q4.txt' u 2:3 w l, '' u 4:5 w l
a=0.05
p [-a:a][-a:a]'e_0.8_q1.txt' u 6:7 w l, '' u 8:9 w l, 'e_0.8_q2.txt' u 6:7 w l, '' u 8:9 w l, 'e_0.8_q3.txt' u 6:7 w l, '' u 8:9 w l, 'e_0.8_q4.txt' u 6:7 w l, '' u 8:9 w l
*/
int main()
{
	try
	{
		//test_calc_ephemeris();
		//test_rtbp2d_trans();
		//test_rtbp2d_transform();
		//test_rtbp2d_calc_energy();
		test_calc_lin_comb();
	}
	catch (const string& msg)
	{
		cerr << "Error: " << msg << endl;
	}

    return 0;
}

#endif
